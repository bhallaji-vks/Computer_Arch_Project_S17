#include "hip/hip_runtime.h"
//CUDA GPU Implementation of Image Smoothing and gradient processing 
//Bhallaji Venkatesan and Divya Sampath Kumar
// Compile by nvcc -arch compute_53 -std=c++11 -I ~/NVIDIA_CUDA-8.0_Samples/common/inc/ -o GPU_imageread_smooth GPU_imageread_smooth.cu

#define _DEFINE_DEPRECATED_HASH_CLASSES 0
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <helper_cuda_drvapi.h>
#include <helper_functions.h>
#include <helper_image.h>
#include <hip/hip_vector_types.h>
#include <helper_string.h>
#include <helper_timer.h>
//#include "Convolution.h"




#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <assert.h>
#include <string.h>
#include <errno.h>
#include <fcntl.h>
#include <unistd.h>
#include <iostream>
#include <cmath>
#include <limits>
#include <sstream>
#include <hash_set>
#include <unordered_set>
#include <iterator>
#include <vector>
#include <chrono>
#define STRONG_EDGE 0xFF
#define NON_EDGE 0x0

#define highThreshold 100.0
#define lowThreshold 90.0
#include "bmp.h"

using namespace std;


std::unordered_set<unsigned int> visitedPixels;

char *BMPInFile = "Cameraman256.bmp";
char *BMPOutFile = "suppress.bmp";


//computeImageGradient();

//suppressNonmaximums();



//texture<float, 2, hipReadModeElementType> deviceMatrixTexture;
texture<unsigned char, 2, hipReadModeElementType> deviceMatrixTexture;
__device__ __constant__ float deviceXGradientMask[9] = {
	-1, 0, 1,
	-2, 0, 2,
	-1, 0, 1
};
__device__ __constant__ float deviceYGradientMask[9] = {
	 1,  2,  1,
	 0,  0,  0,
	-1, -2, -1
};
__device__ __constant__ float deviceGaussianFilterMask[25] ={
	2,  4,  5,  4, 2,
	4,  9, 12,  9, 4,
	5, 12, 15, 12, 5,
	4,  9, 12,  9, 4,
	2,  4,  5,  4, 2
} ;



__global__ void deviceGaussianConvolution(unsigned char * output, int matrixWidth)
{
	int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outputColumn = blockIdx.x * blockDim.x + threadIdx.x;
 
	float accumulator = 0.0;

#pragma unroll
	for(int i = -2; i <= 2; ++i)
	{
		unsigned matrixColumn = outputColumn + i;
#pragma unroll
		for(int j = -2; j <= 2; ++j)
		{
			accumulator += deviceGaussianFilterMask[(2 + i) + (2 + j)] * tex2D(deviceMatrixTexture, matrixColumn, outputRow + j);
		}
	}
	__syncthreads();
	output[outputRow * matrixWidth + outputColumn] = accumulator / 159;
}


__global__ void deviceComputeGradient(unsigned char* outputGradient, unsigned matrixWidth, unsigned int* outputEdgeDirectionClassifications)
{
	int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outputColumn = blockIdx.x * blockDim.x + threadIdx.x;


	// Get gradients
	float xAccumulator = 0.0;
	float yAccumulator = 0.0;

#pragma unroll
	for(int i = -1; i <= 1; ++i)
	{
		unsigned matrixColumn = outputColumn + i;
#pragma unroll
		for(int j = -1; j <= 1; ++j)
		{
			int maskIndex = (1 + i)* 3 + (1 + j);
			//printf("%f\n", tex2D(deviceMatrixTexture, matrixColumn, outputRow + j));
			xAccumulator += deviceXGradientMask[maskIndex] * tex2D(deviceMatrixTexture, matrixColumn, outputRow + j);
			yAccumulator += deviceYGradientMask[maskIndex] * tex2D(deviceMatrixTexture, matrixColumn, outputRow + j);
		}
	}

	int matrixIndex = outputRow * matrixWidth + outputColumn;
	
	// Get gradient magnitude
	outputGradient[matrixIndex] = abs(xAccumulator) + abs(yAccumulator);

	// Determine edge direction
	float edgeDirection = atan2(yAccumulator, xAccumulator) * (180 / 3.14159265) + 180.0;

	// Classify edge directions
	if((edgeDirection >= 22.5 && edgeDirection < 67.5) || (edgeDirection >= 202.5 && edgeDirection < 247.5))
	{
		outputEdgeDirectionClassifications[matrixIndex] = 1;
	}
	else if((edgeDirection >= 67.5 && edgeDirection < 112.5) || (edgeDirection >= 247.5 && edgeDirection < 292.5))
	{
		outputEdgeDirectionClassifications[matrixIndex] = 2;
	}
	else if((edgeDirection >= 112.5 && edgeDirection < 157.5) || (edgeDirection >= 292.5 && edgeDirection < 337.5))
	{
		outputEdgeDirectionClassifications[matrixIndex] = 3;
	}
	else
	{
		outputEdgeDirectionClassifications[matrixIndex] = 0;
	}






}

__global__ void devicesuppressNonmaximums(int width, unsigned int* edgeDirectionClassifications, unsigned char* gradient, int imgsize)
{
	int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outputColumn = blockIdx.x * blockDim.x + threadIdx.x;
	int i= outputRow*width + outputColumn;
	
	int clockwisePerpendicularI;
	int clockwisePerpendicularJ; 
	int clockwisePerpendicularIndex;
	int counterClockwisePerpendicularIndex;

	switch(edgeDirectionClassifications[i])
	{
		case 0:
			clockwisePerpendicularI = outputRow - 1;
			clockwisePerpendicularJ = outputColumn;
			break;
		case 1:
			clockwisePerpendicularI = outputRow - 1;
			clockwisePerpendicularJ = outputColumn + 1;
			break;
		case 2:
			clockwisePerpendicularI = outputRow;
			clockwisePerpendicularJ = outputColumn + 1;
			break;
		case 3:
			clockwisePerpendicularI = outputRow + 1;
			clockwisePerpendicularJ = outputColumn + 1;
			break;
	}
	
	if(clockwisePerpendicularI < 0 || clockwisePerpendicularJ < 0 || clockwisePerpendicularI >= width || clockwisePerpendicularJ >= width)
	{
		clockwisePerpendicularIndex = -1;
	}
	else
	{
		clockwisePerpendicularIndex= clockwisePerpendicularI * width + clockwisePerpendicularJ;
	}
	
	int counterClockwisePerpendicularI;
	int counterClockwisePerpendicularJ;
	
	switch(edgeDirectionClassifications[i])
	{
		case 0:
			counterClockwisePerpendicularI = outputRow + 1;
			counterClockwisePerpendicularJ = outputColumn;
			break;
		case 1:
			counterClockwisePerpendicularI = outputRow + 1;
			counterClockwisePerpendicularJ = outputColumn - 1;
			break;
		case 2:
			counterClockwisePerpendicularI = outputRow;
			counterClockwisePerpendicularJ = outputColumn - 1;
			break;
		case 3:
			counterClockwisePerpendicularI = outputRow - 1;
			counterClockwisePerpendicularJ = outputColumn - 1;
			break;
	}


	if(counterClockwisePerpendicularI < 0 || counterClockwisePerpendicularJ < 0 ||counterClockwisePerpendicularJ >= width || counterClockwisePerpendicularJ >= width)
	{
		counterClockwisePerpendicularIndex = -1;
	}
	else
	{
		counterClockwisePerpendicularIndex= counterClockwisePerpendicularI * width + counterClockwisePerpendicularJ;
	}
	
	float clockwisePerpendicularValue;
	float counterClockwisePerpendicularValue;

	if(clockwisePerpendicularIndex == -1)
	{
		clockwisePerpendicularValue = 0;
	}
	else
	{
		clockwisePerpendicularValue = gradient[clockwisePerpendicularIndex];
				
	}	

			
	if(counterClockwisePerpendicularIndex == -1)
	{
		counterClockwisePerpendicularValue = 0;
	}
	else
	{
		if(counterClockwisePerpendicularIndex < imgsize && counterClockwisePerpendicularIndex >= 0)
		{
			counterClockwisePerpendicularValue = gradient[counterClockwisePerpendicularIndex];
		}
	}
			
	if(gradient[i] <= clockwisePerpendicularValue || gradient[i] <= counterClockwisePerpendicularValue)
	{
		//cout << "\tPixel suppressed." << endl;
		gradient[i] = 0;
			
	}
	else
	{
		//cout << "\tPixel retained." << endl;

	}
	
}

/*
__host__ __device__ void visitNeighbors(int i, int j, unsigned char* gradientImage, unsigned char* outputEdges, int width, int imgsize)
{
	int pixelIndex = i * width + j;
        
	if(i == 0 || j == 0 || i == width - 1 || j == width - 1 || visitedPixels.find(pixelIndex) != visitedPixels.end()  ||gradientImage[pixelIndex] <  lowThreshold)
	{

		visitedPixels.insert(pixelIndex);
		return;
	}

	outputEdges[pixelIndex] =STRONG_EDGE;
	visitedPixels.insert(pixelIndex);

	visitNeighbors(i - 1, j - 1, gradientImage, outputEdges, width, imgsize);
	visitNeighbors(i - 1, j,gradientImage, outputEdges, width, imgsize);
	visitNeighbors(i - 1, j + 1, gradientImage, outputEdges, width, imgsize);
	visitNeighbors(i, j + 1, gradientImage, outputEdges, width, imgsize);
	visitNeighbors(i + 1, j + 1, gradientImage, outputEdges, width, imgsize);
	visitNeighbors(i + 1, j, gradientImage, outputEdges, width, imgsize);
	visitNeighbors(i + 1, j - 1, gradientImage, outputEdges, width, imgsize);
	visitNeighbors(i, j - 1, gradientImage, outputEdges, width, imgsize);
} */


__global__ void devicePerformHysteresis(unsigned char* gradientImage, unsigned char* outputEdges, int width, int imgsize) 
{	
	
	int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outputColumn = blockIdx.x * blockDim.x + threadIdx.x;
	int i= outputRow*width + outputColumn;

	// Mark out borders and all pixels below the high threshold.
	if(gradientImage[i] >= highThreshold)
	{
		//visitedPixels.insert(i);
		outputEdges[i] = STRONG_EDGE;
		//visitNeighbors(outputRow, outputColumn, gradientImage, outputEdges, width, imgsize);
	}
}


void computeGradient(unsigned char * inputMatrix, int matrixWidth, unsigned char * outputGradient, unsigned int* outputEdgeDirections, float *time_smooth, float *time_gradient, float *time_suppression, float *time_hysteresis)
{
	// Create timer.
    //unsigned int timer = 0;
    //CUT_SAFE_CALL(cutCreateTimer(&timer));
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Compute memory sizes.
	int matrixMemorySize = matrixWidth * matrixWidth * sizeof(unsigned char);
	
	// Set up device arrays.
	hipArray* deviceMatrixArray = NULL;
	unsigned char* deviceGradient = NULL;
	unsigned int* deviceEdgeDirections = NULL;
	unsigned char* deviceHysOutput = NULL;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
	hipMallocArray(&deviceMatrixArray, &channelDesc, matrixWidth, matrixWidth);
	hipMalloc((void**)&deviceGradient, matrixMemorySize);
	hipMalloc((void**)&deviceHysOutput, matrixMemorySize);
	hipMalloc((void**)&deviceEdgeDirections, matrixWidth * matrixWidth * sizeof(unsigned int));

	// Copy inputs to device.
	hipMemcpyToArray(deviceMatrixArray, 0, 0, inputMatrix, matrixMemorySize, hipMemcpyHostToDevice);

	// Set up image matrix as a texture.
	deviceMatrixTexture.addressMode[0] = hipAddressModeClamp;
	deviceMatrixTexture.addressMode[1] = hipAddressModeClamp;
	hipBindTextureToArray(deviceMatrixTexture, deviceMatrixArray);

	// Start timer.
	//CUT_SAFE_CALL(cutStartTimer(timer));

	// Do it!
	dim3 dimGrid(matrixWidth / 16, matrixWidth / 16);
	dim3 dimBlock(16, 16);
	hipEventRecord(start);
	deviceGaussianConvolution<<<dimGrid, dimBlock>>>(deviceGradient, matrixWidth);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(time_smooth, start, stop);
	hipMemcpy(outputGradient, deviceGradient, matrixMemorySize, hipMemcpyDeviceToHost);
	hipUnbindTexture(deviceMatrixTexture);
	hipMemcpyToArray(deviceMatrixArray, 0, 0, outputGradient, matrixMemorySize, hipMemcpyHostToDevice);
	hipBindTextureToArray(deviceMatrixTexture, deviceMatrixArray);



	hipEventRecord(start);
	deviceComputeGradient<<<dimGrid, dimBlock>>>(deviceGradient, matrixWidth, deviceEdgeDirections);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(time_gradient, start, stop);
	// Check for errors.
	//CUT_CHECK_ERROR("Kernel execution failed!");

	// Copy device result to host.
	hipMemcpy(outputGradient, deviceGradient, matrixMemorySize, hipMemcpyDeviceToHost);
	hipMemcpy(outputEdgeDirections, deviceEdgeDirections, matrixMemorySize, hipMemcpyDeviceToHost);	
	
	hipEventRecord(start);
	devicesuppressNonmaximums<<<dimGrid, dimBlock>>>(matrixWidth, deviceEdgeDirections, deviceGradient, matrixMemorySize);
	hipMemcpy(outputGradient, deviceGradient, matrixMemorySize, hipMemcpyDeviceToHost);	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(time_suppression, start, stop);
	
	hipEventRecord(start);
	devicePerformHysteresis<<<dimGrid, dimBlock>>>(deviceGradient,deviceHysOutput, matrixWidth, matrixMemorySize);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(time_hysteresis, start, stop);
	hipMemcpy(outputGradient, deviceHysOutput, matrixMemorySize, hipMemcpyDeviceToHost);

	hipFreeArray(deviceMatrixArray);
	hipFree(deviceGradient);
	hipFree(deviceEdgeDirections);
	hipUnbindTexture(deviceMatrixTexture);

}




void BitMapRead(char *file,struct bmp_header *bmp, struct dib_header *dib, unsigned char **data, unsigned char **palete)
{
	
	size_t palete_size;

	int fd;

	if((fd = open(file, O_RDONLY)) <0)
		FATAL("Open Source");
	if(read(fd, bmp, BMP_SIZE) != BMP_SIZE)
		FATAL("Read BMP Header");
	if(read(fd, dib, DIB_SIZE) != DIB_SIZE)
		FATAL("Read DIB Header");
	assert(dib->bpp ==8);	

	palete_size = bmp->offset - BMP_SIZE - DIB_SIZE;
	if(palete_size > 0) {
		*palete = (unsigned char *)malloc(palete_size);
		int go = read(fd, *palete, palete_size);
		if(go != palete_size) {
			FATAL("Read Palete");
		}	
	}

	*data = (unsigned char *)malloc(dib->image_size);
	if(read(fd, *data, dib->image_size) != dib->image_size)
		//close(fd);
		FATAL("Read Image");
	close(fd);
}

void BitMapWrite(char *file, struct bmp_header *bmp, struct dib_header *dib, unsigned char *data, unsigned char *palete)
{
	size_t palete_size;
	int fd;

	palete_size = bmp->offset - BMP_SIZE - DIB_SIZE;
	

	if((fd = open(file, O_WRONLY | O_CREAT | O_TRUNC,S_IRUSR | S_IWUSR | S_IRGRP)) <0)
		FATAL("Open Destination");

	if(write(fd, bmp, BMP_SIZE) != BMP_SIZE)
		FATAL("Write BMP Header");

	if(write(fd, dib, DIB_SIZE) != DIB_SIZE)
		FATAL("Write DIB Header");
	
	if(palete_size != 0) {
		if(write(fd, palete, palete_size) != palete_size)
			FATAL("Write Palete");
	}

	
	if(write(fd, data, dib->image_size) != dib->image_size)
		FATAL("Write Image");
	

	
	close(fd);

}

void ParseArguments(int argc, char** argv)
{
	for (int i =0; i<argc; ++i)
	{
		if(strcmp(argv[i],"--file") == 0 || strcmp(argv[i],"-file") == 0) {
			BMPInFile = argv[i+1];
			i = i+1;
		}

		if(strcmp(argv[i],"--out") == 0 || strcmp(argv[i],"-out") == 0) {
			BMPOutFile = argv[i+1];
			i = i+1;
		}
	}
}


int main(int argc, char** argv)
{
 	ParseArguments(argc, argv);
//void computeGradient(const float* inputMatrix, int matrixWidth, float* outputGradient)
//deviceGaussianConvolution<<<dimGrid, dimBlock>>>(deviceGradient, matrixWidth);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	struct bmp_header bmp;
	struct dib_header dib;
	unsigned char *palete = NULL;
	unsigned char *data = NULL;
	unsigned char *out = NULL;
	unsigned int *edgeDirectionClassifications = NULL;
	float time_smooth = 0;
	float time_gradient = 0;
	float time_suppression = 0;
	float time_hysteresis = 0;

	BitMapRead(BMPInFile, &bmp, &dib, &data, &palete);
	out = (unsigned char *)malloc(dib.image_size);
	edgeDirectionClassifications = (unsigned int *)malloc(dib.image_size);
	hipEventRecord(start);
	//Gaussian Smoothening
        auto begin2 = chrono::high_resolution_clock::now();                            
	computeGradient(data, dib.width, out,edgeDirectionClassifications, &time_smooth, &time_gradient, &time_suppression, &time_hysteresis);
	//convolution(data, out, dib.width, gaussianMask, 5, gaussianMaskWeight);
	BitMapWrite("GPU_Gaussian_Smooth_Gradient_suppression2.bmp", &bmp, &dib, out, palete);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	data = NULL;
	free(data);
	//suppressNonmaximums(dib.width, edgeDirectionClassifications, out, dib.image_size);
	//BitMapWrite("GPU_Gaussian_Smooth_Gradient_suppression.bmp", &bmp, &dib, out, palete);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Input Image size %d x %d pixels \n", dib.height, dib.width); 
	printf("Elapsed Time for smoothing:%f ms\n",time_smooth);
	printf("Elapsed Time for gradient:%f ms\n",time_gradient);
	printf("Elapsed Time for suppression:%f ms\n",time_suppression);
	printf("Elapsed Time for hysteresis:%f ms\n",time_hysteresis);
	printf("Elapsed Time for total edge detection:%f ms\n",milliseconds);
	

}


			

			

