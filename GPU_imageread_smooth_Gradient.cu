#include "hip/hip_runtime.h"
//CUDA GPU Implementation of Image Smoothing and gradient processing 
//Bhallaji Venkatesan and Divya Sampath Kumar
// Compile by nvcc -arch compute_53 -std=c++11 -I ~/NVIDIA_CUDA-8.0_Samples/common/inc/ -o GPU_imageread_smooth GPU_imageread_smooth.cu

#define _DEFINE_DEPRECATED_HASH_CLASSES 0
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <helper_cuda_drvapi.h>
#include <helper_functions.h>
#include <helper_image.h>
#include <hip/hip_vector_types.h>
#include <helper_string.h>
#include <helper_timer.h>
//#include "Convolution.h"




#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <assert.h>
#include <string.h>
#include <errno.h>
#include <fcntl.h>
#include <unistd.h>
#include <iostream>
#include <cmath>
#include <limits>
#include <sstream>
#include <hash_set>
#include <unordered_set>
#include <iterator>
#include <vector>
#define STRONG_EDGE 0xFF
#define NON_EDGE 0x0
#include "bmp.h"

char *BMPInFile = "car.bmp";
char *BMPOutFile = "suppress.bmp";


//computeImageGradient();

//suppressNonmaximums();



//texture<float, 2, hipReadModeElementType> deviceMatrixTexture;
texture<unsigned char, 2, hipReadModeElementType> deviceMatrixTexture;
__device__ __constant__ float deviceXGradientMask[9] = {
	-1, 0, 1,
	-2, 0, 2,
	-1, 0, 1
};
__device__ __constant__ float deviceYGradientMask[9] = {
	 1,  2,  1,
	 0,  0,  0,
	-1, -2, -1
};
__device__ __constant__ float deviceGaussianFilterMask[25] ={
	2,  4,  5,  4, 2,
	4,  9, 12,  9, 4,
	5, 12, 15, 12, 5,
	4,  9, 12,  9, 4,
	2,  4,  5,  4, 2
} ;



__global__ void deviceGaussianConvolution(unsigned char * output, int matrixWidth)
{
	int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outputColumn = blockIdx.x * blockDim.x + threadIdx.x;
 
	float accumulator = 0.0;

#pragma unroll
	for(int i = -2; i <= 2; ++i)
	{
		unsigned matrixColumn = outputColumn + i;
#pragma unroll
		for(int j = -2; j <= 2; ++j)
		{
			accumulator += deviceGaussianFilterMask[(2 + i) + (2 + j)] * tex2D(deviceMatrixTexture, matrixColumn, outputRow + j);
		}
	}
	__syncthreads();
	output[outputRow * matrixWidth + outputColumn] = accumulator / 159;
}


__global__ void deviceComputeGradient(unsigned char* outputGradient, unsigned matrixWidth)
{
	int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outputColumn = blockIdx.x * blockDim.x + threadIdx.x;


	// Get gradients
	float xAccumulator = 0.0;
	float yAccumulator = 0.0;

#pragma unroll
	for(int i = -1; i <= 1; ++i)
	{
		unsigned matrixColumn = outputColumn + i;
#pragma unroll
		for(int j = -1; j <= 1; ++j)
		{
			int maskIndex = (1 + i)* 3 + (1 + j);
			//printf("%f\n", tex2D(deviceMatrixTexture, matrixColumn, outputRow + j));
			xAccumulator += deviceXGradientMask[maskIndex] * tex2D(deviceMatrixTexture, matrixColumn, outputRow + j);
			yAccumulator += deviceYGradientMask[maskIndex] * tex2D(deviceMatrixTexture, matrixColumn, outputRow + j);
		}
	}

	int matrixIndex = outputRow * matrixWidth + outputColumn;
	
	// Get gradient magnitude
	outputGradient[matrixIndex] = abs(xAccumulator) + abs(yAccumulator);
}


void computeGradient(unsigned char * inputMatrix, int matrixWidth, unsigned char * outputGradient)
{
	// Create timer.
    //unsigned int timer = 0;
    //CUT_SAFE_CALL(cutCreateTimer(&timer));

	// Compute memory sizes.
	int matrixMemorySize = matrixWidth * matrixWidth * sizeof(unsigned char);
	
	// Set up device arrays.
	hipArray* deviceMatrixArray = NULL;
	unsigned char* deviceGradient = NULL;
	//unsigned* deviceEdgeDirections = NULL;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
	hipMallocArray(&deviceMatrixArray, &channelDesc, matrixWidth, matrixWidth);
	hipMalloc((void**)&deviceGradient, matrixMemorySize);
	//hipMalloc((void**)&deviceEdgeDirections, matrixWidth * matrixWidth * sizeof(unsigned));

	// Copy inputs to device.
	hipMemcpyToArray(deviceMatrixArray, 0, 0, inputMatrix, matrixMemorySize, hipMemcpyHostToDevice);

	// Set up image matrix as a texture.
	deviceMatrixTexture.addressMode[0] = hipAddressModeClamp;
	deviceMatrixTexture.addressMode[1] = hipAddressModeClamp;
	hipBindTextureToArray(deviceMatrixTexture, deviceMatrixArray);

	// Start timer.
	//CUT_SAFE_CALL(cutStartTimer(timer));

	// Do it!
	dim3 dimGrid(matrixWidth / 16, matrixWidth / 16);
	dim3 dimBlock(16, 16);

	deviceGaussianConvolution<<<dimGrid, dimBlock>>>(deviceGradient, matrixWidth);
	hipMemcpy(outputGradient, deviceGradient, matrixMemorySize, hipMemcpyDeviceToHost);
	hipUnbindTexture(deviceMatrixTexture);
	hipMemcpyToArray(deviceMatrixArray, 0, 0, outputGradient, matrixMemorySize, hipMemcpyHostToDevice);
	hipBindTextureToArray(deviceMatrixTexture, deviceMatrixArray);




	deviceComputeGradient<<<dimGrid, dimBlock>>>(deviceGradient, matrixWidth);

	// Check for errors.
	//CUT_CHECK_ERROR("Kernel execution failed!");

	// Copy device result to host.
	hipMemcpy(outputGradient, deviceGradient, matrixMemorySize, hipMemcpyDeviceToHost);
	//hipMemcpy(outputEdgeDirections, deviceEdgeDirections, matrixMemorySize, hipMemcpyDeviceToHost);	


	hipFreeArray(deviceMatrixArray);
	hipFree(deviceGradient);
	//hipFree(deviceEdgeDirections);
	hipUnbindTexture(deviceMatrixTexture);

}




void BitMapRead(char *file,struct bmp_header *bmp, struct dib_header *dib, unsigned char **data, unsigned char **palete)
{
	
	size_t palete_size;

	int fd;

	if((fd = open(file, O_RDONLY)) <0)
		FATAL("Open Source");
	if(read(fd, bmp, BMP_SIZE) != BMP_SIZE)
		FATAL("Read BMP Header");
	if(read(fd, dib, DIB_SIZE) != DIB_SIZE)
		FATAL("Read DIB Header");
	assert(dib->bpp ==8);	

	palete_size = bmp->offset - BMP_SIZE - DIB_SIZE;
	if(palete_size > 0) {
		*palete = (unsigned char *)malloc(palete_size);
		int go = read(fd, *palete, palete_size);
		if(go != palete_size) {
			FATAL("Read Palete");
		}	
	}

	*data = (unsigned char *)malloc(dib->image_size);
	if(read(fd, *data, dib->image_size) != dib->image_size)
		//close(fd);
		FATAL("Read Image");
	close(fd);
}

void BitMapWrite(char *file, struct bmp_header *bmp, struct dib_header *dib, unsigned char *data, unsigned char *palete)
{
	size_t palete_size;
	int fd;

	palete_size = bmp->offset - BMP_SIZE - DIB_SIZE;
	

	if((fd = open(file, O_WRONLY | O_CREAT | O_TRUNC,S_IRUSR | S_IWUSR | S_IRGRP)) <0)
		FATAL("Open Destination");

	if(write(fd, bmp, BMP_SIZE) != BMP_SIZE)
		FATAL("Write BMP Header");

	if(write(fd, dib, DIB_SIZE) != DIB_SIZE)
		FATAL("Write DIB Header");
	
	if(palete_size != 0) {
		if(write(fd, palete, palete_size) != palete_size)
			FATAL("Write Palete");
	}

	
	if(write(fd, data, dib->image_size) != dib->image_size)
		FATAL("Write Image");
	

	
	close(fd);

}


int main()
{

//void computeGradient(const float* inputMatrix, int matrixWidth, float* outputGradient)
//deviceGaussianConvolution<<<dimGrid, dimBlock>>>(deviceGradient, matrixWidth);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	struct bmp_header bmp;
	struct dib_header dib;
	unsigned char *palete = NULL;
	unsigned char *data = NULL;
	unsigned char *out = NULL;

	BitMapRead(BMPInFile, &bmp, &dib, &data, &palete);
	out = (unsigned char *)malloc(dib.image_size);
	
	hipEventRecord(start);
	//Gaussian Smoothening
	computeGradient(data, dib.width, out);
	//convolution(data, out, dib.width, gaussianMask, 5, gaussianMaskWeight);
	BitMapWrite("GPU_Gaussian_Smooth_Gradient.bmp", &bmp, &dib, out, palete);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Elapsed Time for smoothing:%f\n",milliseconds);

}

